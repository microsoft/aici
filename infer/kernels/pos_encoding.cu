// adapted from vllm pos_encoding_kernels.cu


#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

template<typename scalar_t, bool IS_NEOX>
inline __device__ void apply_rotary_embedding(
  scalar_t* __restrict__ arr,
  const scalar_t* __restrict__ cos_ptr,
  const scalar_t* __restrict__ sin_ptr,
  int rot_offset,
  int embed_dim)
{
  int x_index, y_index;
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = __ldg(cos_ptr + x_index);
    sin = __ldg(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = __ldg(cos_ptr + x_index / 2);
    sin = __ldg(sin_ptr + x_index / 2);
  }

  const scalar_t x = arr[x_index];
  const scalar_t y = arr[y_index];
  arr[x_index] = x * cos - y * sin;
  arr[y_index] = y * cos + x * sin;
}

template<typename scalar_t, bool IS_NEOX>
__global__ void rotary_embedding_kernel(
  const int64_t* __restrict__ positions,        // [num_tokens]
  scalar_t* __restrict__ query,                 // [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key,                   // [num_tokens, num_kv_heads, head_size]
  const scalar_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int query_stride,
  const int key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const scalar_t* cos_ptr = cache_ptr;
  const scalar_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

#define scalar_t __hip_bfloat16

extern "C" void rotary_embedding_bf16(
  const int64_t* __restrict__ positions,        // [num_tokens]
  scalar_t* __restrict__ query,                 // [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key,                   // [num_tokens, num_kv_heads, head_size]
  const scalar_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int32_t num_tokens,
  const int32_t rot_dim,
  const int32_t query_stride,
  const int32_t key_stride,
  const int32_t num_heads,
  const int32_t num_kv_heads,
  const int32_t head_size)
{

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * rot_dim / 2, 512));
  const hipStream_t stream = 0; // Use the default stream.
  // const cudaStream_t stream = at::cuda::getCurrentCUDAStream();
  rotary_embedding_kernel<scalar_t, true><<<grid, block, 0, stream>>>(
    positions, query, key, cos_sin_cache, rot_dim, query_stride, key_stride, num_heads, num_kv_heads, head_size);
}

#if 0

void rotary_embedding(
  torch::Tensor& positions,         // [num_tokens]
  torch::Tensor& query,             // [num_tokens, num_heads * head_size]
  torch::Tensor& key,               // [num_tokens, num_kv_heads * head_size]
  int head_size,
  torch::Tensor& cos_sin_cache,     // [max_position, rot_dim]
  bool is_neox) {
  int num_tokens = query.size(0);
  int rot_dim = cos_sin_cache.size(1);
  int num_heads = query.size(1) / head_size;
  int num_kv_heads = key.size(1) / head_size;
  int query_stride = query.stride(0);
  int key_stride = key.stride(0);

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * rot_dim / 2, 512));
  const cudaStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
    query.scalar_type(),
    "rotary_embedding",
    [&] {
      if (is_neox) {
        vllm::rotary_embedding_kernel<scalar_t, true><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(),
          query.data_ptr<scalar_t>(),
          key.data_ptr<scalar_t>(),
          cos_sin_cache.data_ptr<scalar_t>(),
          rot_dim,
          query_stride,
          key_stride,
          num_heads,
          num_kv_heads,
          head_size);
      } else {
        vllm::rotary_embedding_kernel<scalar_t, false><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(),
          query.data_ptr<scalar_t>(),
          key.data_ptr<scalar_t>(),
          cos_sin_cache.data_ptr<scalar_t>(),
          rot_dim,
          query_stride,
          key_stride,
          num_heads,
          num_kv_heads,
          head_size);
      }
    });
}

#endif